#include "hip/hip_runtime.h"
#include "main_Kernel.h"

#include <iostream>

#include <stb/stb_image_write.h>
#include <glm/glm.hpp>
#include <easylogging/easylogging++.h>

#include <hip/hip_runtime.h>
#include <>


__global__ void kernel(int width, int height, glm::vec3* image) {
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid >= width * height) return;

	int y = gid / width;
	int x = gid % width;

	float r = x / (width - 1.0f);
	float g = y / (height - 1.0f);
	float b = (r + g) / 2.0f;

	image[gid][0] = r;
	image[gid][1] = g;
	image[gid][2] = b;
}

Renderer_cu::Renderer_cu(int width, int height) : width(width), height(height) {
	LOG(INFO) << "Allocating image memory on device";
	hipMalloc((void**)&d_image, width * height * sizeof(glm::vec3));
	LOG(INFO) << "allocation finished";
}

void Renderer_cu::Run() {

	int threads = 32;
	int blocks = (width * height + threads - 1) / threads;

	LOG(INFO) << "Launching render kernel";
	kernel<<<blocks, threads>>>(width, height, d_image);
	hipDeviceSynchronize();
	LOG(INFO) << "kernel finished";
}

std::vector<glm::vec3> Renderer_cu::Download() {

	LOG(INFO) << "Downloading kernel image from device";
	std::vector<glm::vec3> h_image(width * height, glm::vec3(0.0f));
	hipMemcpy((glm::vec3*)h_image.data(), d_image, width * height * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	LOG(INFO) << "download done";

	return h_image;
}

void Renderer_cu::Delete() {

	LOG(INFO) << "Deleting kernel device memory";
	if (d_image == nullptr) LOG(WARNING) << "Attempting to free after free device memory";

	hipFree(d_image);
	d_image = nullptr;

	LOG(INFO) << "deletion finished";
}